/* exffunction.cpp
 * 
 * not a stand-alone! This file is sourced
 * from 
 * which compiles it into an R function
 *
 * Computing the ExF requires knowledge of the local
 network topology. We use R/Igraph to extract the
 node neighborhood. The code here parses this R data
 into a C++ representation, then uses this 
 representation to determine the ExF. Hence I divide
 the code into "helper functions" which build this
 representation, and the main function "ExF" which
 provides the interface to R.
 
 The graph is stored as an indexed edgelist. The
 index tracks the start/end of each node in the 
 "ego" portion of the edgelist.

 I get cleaner code by using a FULL edge list, 
 i.e.  containing both 1--2 and 2--1, then by
 a more compact list which implies bi-directionality;
 a compact list would probably be faster.
 
 Also, I am not sure if creating the edge index
 saves any time. Since the edge list is assumed
 to be sorted, it may be faster to binary search
 the edge list for the start positions, than to
 create/store the index. Again, however, this 
 creates more complex code.
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <algorithm>
#include <vector>
#include <queue>
#include <map>
#include <iostream>
#include <iterator> // for ostream_iterator, used for debugging output
 
typedef std::vector<int> svi;
typedef std::vector<int>::iterator svii;
typedef std::map<int,int> smi;
typedef std::map<int,int>::iterator smii;

//////////////////////////////////////////////////////////////////////
// HELPER FUNCTIONS 
// set_egostarts: Creates a node-based index to the edgelist
// get_neighbors: Finds neighbors from a seed at distance 1 and 2
// cluster_degree: Returns the degree of a given cluster.
//////////////////////////////////////////////////////////////////////


/* Given a reference to the index, and a 
   SORTED, FULL edgelist, create the index 
   and return 0 (success).

   @param[out] egostart, egoend. The index
   @param[in] egos, alters. The edgelist
   @return success code, in case we later
   wish to add a safety check
 */
int set_egostarts(svi & egostart, svi & egoend,
		  const svi & egos,  const svi & alters){
  egostart.clear(); egoend.clear();
  int nnum=0,cnt=0;
  egostart.push_back(cnt);
  while(nnum<egos[0]){ // no edge
    egostart.push_back(cnt);
    egoend.push_back(cnt);
    nnum++;
  } 
  for(unsigned int i=1;i<egos.size();i++){
    cnt++;
    if(egos[i] != egos[i-1]){
      //std::cout<<egos[i]<<"  ";
      egostart.push_back(cnt); 
      egoend.push_back(cnt);
      nnum++;
      while(nnum<egos[i]){ // no edge
	//std::cout<<"*";
	nnum++;
      }}
  }
  egoend.push_back(++cnt);
  return 0;
}


/* Find the neighbors at distance one and two
   from a seed in a network.

   @param[out] dOne, dTwo. References to the vectors which 
   store the neighbor node ids
   @param[in] seed. The seed node
   @param[in] egostart,egoend, alters. The needed network topology
   @return success (usefull for debugging, or adding safety checks)
*/
int get_neighbors(svi & dOne, svi & dTwo,
		  const int seed,
		  const svi & egostart, const svi & egoend,
		  const svi & alters){
  dOne.clear(); dTwo.clear();  
  std::queue<int> q;
  smi visited; // maps, as I index by node #
  smi distance; 
  int i,curN,curD;
    
  // initialize the queue
  q.push(seed);
  visited[seed]=1;
  distance[seed]=0;
  //std::cout<<egostart.size() << std::endl;
  while(! q.empty()){ // run the BFS
    curN=q.front(); q.pop();  // grab the next node 
    //std::cout<<curN<<"  "<<distance[curN]<<std::endl;
    switch(distance[curN]){ // store the results, or exit if done
    case 1: dOne.push_back(curN); break;
    case 2: dTwo.push_back(curN); break;
    case 3: return 0; // STOP CONDITION (first node at dist 3
    default: break; // should hit this for the seed node, dist 0      
    }
    for(i=egostart[curN];i<egoend[curN];i++){ // add its children to the queue
      curD=alters[i];
      if(! visited[curD]){
	visited[curD]=1;
	distance[curD]=distance[curN]+1;
	q.push(curD);
      }}
  }
  // the BFS should only get here if it finds no nodes at distance 3
  //std::cout<<"found "<<dOne.size()+dTwo.size()<<" neighbors."<<std::endl;
  //std::cout<<"get_neighbors had a problem."<<std::endl;
  return 0; 
}


/* Find the degree of a cluster

   returned as a double so we don't have to
   cast it later.
*/ 
double cluster_degree(svi & clusterNodes,
		      const svi & egostart, const svi & egoend,
		      const svi & alters) {
  int degree=0;
  // for each element of clusterNodes,
  //   for each neighbor of that element,
  //      which is not in clusterNodes
  //         increase the degree
  for(svii clusteriter=clusterNodes.begin();
      clusteriter!=clusterNodes.end();clusteriter++){
    for(int i=egostart[*clusteriter];i<egoend[*clusteriter];i++){
      if(std::find(clusterNodes.begin(),clusterNodes.end(),alters[i]) == 
	 clusterNodes.end()){
	degree++;
      }}}
  return(degree);
}


////////////////////////////////////////////////////////
// MAIN FUNCTION
//

/* Given a FULL, SORTED edgelist and a seed node,
   return the ExF of the seed.
   It is easy to create a full, sorted edgelist from
   R/IGraph by i.e. 
   elist <- get.edgelist(graph)
   elist <- rbind(elist,elist[,c(2,1)]
   eorder <- order(elist[,1],elist[,2])
   
   Then call:
   exfcpp(elist[eorder,1],elist[eorder,2],seed)

   Note that the R function
   defined in 
   does this for you.
   
   Note also that only the local neighborhood of 
   the seed is needed (and that extracting this 
   using Igraph may change node indexing)
*/

// [[Rcpp::export]]
double exfcpp(svi egosVect, svi altersVect, int seed){
  // SAFETY: check if the seed is in the edgelist!
  // SAFETY: check that the edge list is complete and sorted
  /////////////////////////////////////////////////////////////
  // set up the graph structure 
  svi egostart,egoend; // indexes into egos, alters
  svi dOne, dTwo; // nodes at distance 1 (resp 2) from seed
  int all_ok;
  all_ok = set_egostarts(egostart, egoend, egosVect, altersVect);
  //std::cout<<"egostarts ok "<<all_ok<<std::endl;
  if(all_ok != 0){ return -1;}
  all_ok=get_neighbors(dOne, dTwo, seed, egostart, egoend, altersVect);
  //std::cout<<"get neighbors "<<all_ok<<std::endl;
  if(all_ok != 0){ return -2;}
  /////////////////////////////////////////////////////////////
  // Initialize the vectors and etc to store the FI values 
  // for each cluster
  svi tmp(3); // stores the nodes in the cluster
  tmp[0]=seed;
  svii i,j; // iterators over the neighbors of the seed
  double clustFI=0.0, totalFI=0.0; // cluster FI and total FI
  std::vector<double>  FIvalues; // the vector of FI values
  FIvalues.reserve(10000); // faster to use a constant that to guestimate
  ///////////////////////////////////////////////////////////////
  // Iterate over all possible clusters of size 2 (plus the seed).
  // The iteration is over all nodes at distance one from the source,
  //     within this loop we consider both all remaining dOne nodes
  //     and all dTwo nodes reachable from the current dOne node.
  for(i=dOne.begin();i!=dOne.end();i++){ 
    tmp[1]=*i;
    for(j=i+1;j!=dOne.end();j++){ // the remaining dOne nodes
      tmp[2]=*j;
	  clustFI=cluster_degree(tmp,egostart,egoend,altersVect);
      // add it once for each way the cluster could form
      FIvalues.push_back(clustFI); totalFI+=clustFI;
      FIvalues.push_back(clustFI); totalFI+=clustFI;
      for(int edgeindx=egostart[*i];edgeindx<egoend[*i];edgeindx++){ 
	if(altersVect[edgeindx]==*j){
	  FIvalues.push_back(clustFI); totalFI+=clustFI;
	  FIvalues.push_back(clustFI); totalFI+=clustFI;
	}}
    }
    // now search for all neighbors of i at distance two
    for(int neigh=egostart[*i];neigh<egoend[*i];neigh++){ 
      j=find(dTwo.begin(),dTwo.end(),altersVect[neigh]);
      if(j != dTwo.end()){
	tmp[2]=*j;
	clustFI=cluster_degree(tmp,egostart,egoend,altersVect);
	FIvalues.push_back(clustFI); totalFI+=clustFI;	
      }}
  } // end iteration over all clusters
  /////////////////////////////////////////////////////////////
  // compute "entropy" of the FI values
  double normalizedFI, ExF(0); 
  //std::cout<<"clust degs: ";
  for(std::vector<double>::iterator i=FIvalues.begin();i!=FIvalues.end();i++){
    //std::cout<<*i<<" ";
    normalizedFI =*i/totalFI;
    ExF -= (log(normalizedFI)*normalizedFI);
  }
  //std::cout<<std::endl;
  return(ExF);
}
  







